#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>
#include <stdio.h>

#include "utils/utils.h"
#include "arch.h"
/**
* Fernando Fernandes, 10/2022
* Add the support for ballot_sync for architectures >= 70
*/
__device__ inline int profiler_ballot(int pred) {
#if __CUDA_ARCH__ >= 700
    return  __ballot_sync(0xFFFFFFFF, pred);
#else
    return ballot(pred);
#endif
}

// Global counters are incremented once per warp 
extern "C" __device__ __noinline__ void count_instrs(uint64_t pcounters, int index, int grp_index, int predicate, int num_counters) {    
	uint64_t *counters = (uint64_t*)pcounters;

	// Optimization: Instead of all the threads in a warp performing atomicAdd,
	// let's count the number of active threads with predicate=1 in a warp and let just one thread
	// (leader) in the warp perform the atomicAdd
    const int active_mask = profiler_ballot(1);
	const int leader = __ffs(active_mask) - 1;
	const int laneid = get_laneid();

	// compute the predicate mask 
    const int predicate_mask = profiler_ballot(1); //ballot(predicate);
	const int num_threads = __popc(predicate_mask);

	if (laneid == leader) { // Am I the leader thread
		atomicAdd((unsigned long long *)&counters[index], num_threads);
		atomicAdd((unsigned long long *)&counters[NUM_ISA_INSTRUCTIONS+grp_index], num_threads);
		atomicAdd((unsigned long long *)&counters[num_counters-2], num_threads*(grp_index != G_NODEST));
		atomicAdd((unsigned long long *)&counters[num_counters-1], num_threads*(1 - ((grp_index == G_NODEST) || (grp_index == G_PR))));
	}
}
