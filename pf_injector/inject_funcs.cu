#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <stdint.h>
#include <stdio.h>

#include "nvbit_reg_rw.h"
#include "utils/utils.h"
#include "pf_injector.h"
#include "arch.h"


extern "C" __device__ __noinline__ void inject_error(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int maxRegs) {

				inj_info_t* inj_info = (inj_info_t*)piinfo; 
				uint32_t verbose_device = *((uint32_t *)pverbose_device);

				uint32_t smid;
				asm("mov.u32 %0, %smid;" :"=r"(smid));
				if (smid != inj_info->injSMID) 
								return; // This is not the selected SM. No need to proceed.

				uint32_t laneid;
				asm("mov.u32 %0, %laneid;" :"=r"(laneid));
				if (laneid != inj_info->injLaneID) 
								return; // This is not the selected Lane ID. No need to proceed.

				assert(numDestGPRs > 0);
				uint32_t injAfterVal = 0; 
				uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
				if (DUMMY) {
								injAfterVal = injBeforeVal;
				} else {
								injAfterVal = injBeforeVal ^ inj_info->injMask; 
								nvbit_write_reg(destGPRNum, injAfterVal);
				}
				// updating counter/flag to check whether the error was injected
				if (verbose_device) printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x\n", destGPRNum, injBeforeVal, nvbit_read_reg(destGPRNum), injAfterVal);
				inj_info->errorInjected = true; 
				atomicAdd((unsigned long long*) &inj_info->injNumActivations, 1LL);  
}

